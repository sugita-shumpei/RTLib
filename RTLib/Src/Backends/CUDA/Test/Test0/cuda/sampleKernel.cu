#include "hip/hip_runtime.h"
#include <RTLib/Backends/CUDA/Math/Random.h>
#include <RTLib/Backends/CUDA/Math/VectorFunctions.h>
class Perline {
public:
    static RTLIB_DEVICE auto Generate(float x, float y, float z) -> float
    {
        inline constexpr unsigned char p[512] = {
123,61,55,66,162,185,6,175,34,233,24,13,201,151,89,136,
192,73,103,121,213,78,190,247,105,114,108,163,95,27,132,193,
115,143,180,118,133,255,177,171,60,200,43,110,240,206,199,124,
3,75,65,126,209,140,187,134,97,232,173,179,253,54,183,86,
150,215,84,29,212,48,207,117,197,222,181,148,69,170,92,19,
88,106,56,196,241,226,82,155,71,182,25,22,216,1,35,120,
235,58,244,20,41,122,221,227,62,137,11,246,211,129,63,254,
243,178,141,131,228,116,112,50,30,234,8,94,217,74,204,98,
135,67,229,76,237,10,45,72,158,100,236,149,31,119,40,18,
0,38,15,127,64,85,107,152,109,231,28,195,93,42,159,157,
172,113,81,49,9,218,164,242,79,198,90,16,219,205,37,208,
5,59,2,169,77,32,230,160,238,168,142,147,111,156,186,53,
87,161,248,214,36,154,139,57,251,21,225,210,191,14,184,245,
102,176,202,52,23,165,174,80,138,224,239,104,194,166,249,83,
4,128,99,96,220,44,125,223,33,46,144,26,91,167,51,68,
7,189,146,203,101,145,17,39,153,130,250,70,188,12,47,252,
123,61,55,66,162,185,6,175,34,233,24,13,201,151,89,136,
192,73,103,121,213,78,190,247,105,114,108,163,95,27,132,193,
115,143,180,118,133,255,177,171,60,200,43,110,240,206,199,124,
3,75,65,126,209,140,187,134,97,232,173,179,253,54,183,86,
150,215,84,29,212,48,207,117,197,222,181,148,69,170,92,19,
88,106,56,196,241,226,82,155,71,182,25,22,216,1,35,120,
235,58,244,20,41,122,221,227,62,137,11,246,211,129,63,254,
243,178,141,131,228,116,112,50,30,234,8,94,217,74,204,98,
135,67,229,76,237,10,45,72,158,100,236,149,31,119,40,18,
0,38,15,127,64,85,107,152,109,231,28,195,93,42,159,157,
172,113,81,49,9,218,164,242,79,198,90,16,219,205,37,208,
5,59,2,169,77,32,230,160,238,168,142,147,111,156,186,53,
87,161,248,214,36,154,139,57,251,21,225,210,191,14,184,245,
102,176,202,52,23,165,174,80,138,224,239,104,194,166,249,83,
4,128,99,96,220,44,125,223,33,46,144,26,91,167,51,68,
7,189,146,203,101,145,17,39,153,130,250,70,188,12,47,252,
        };
        x = fmodf(x, 256.0f);
        y = fmodf(y, 256.0f);
        z = fmodf(z, 256.0f);
        int xi = static_cast<int>(floorf(x));
        int yi = static_cast<int>(floorf(y));
        int zi = static_cast<int>(floorf(z));
        float xf = x - static_cast<float>(xi);
        float yf = y - static_cast<float>(yi);
        float zf = z - static_cast<float>(zi);
        int aaa = p[p[p[xi+0]+yi+0]+zi+0];
        int bbb = p[p[p[xi+1]+yi+1]+zi+1];
        int baa = p[p[p[xi+1]+yi+0]+zi+0];
        int aba = p[p[p[xi+0]+yi+1]+zi+0];
        int aab = p[p[p[xi+0]+yi+0]+zi+1];
        int abb = p[p[p[xi+0]+yi+1]+zi+1];
        int bab = p[p[p[xi+1]+yi+0]+zi+1];
        int bba = p[p[p[xi+1]+yi+1]+zi+0];
        float u = Fade(xf);
        float v = Fade(yf);
        float w = Fade(zf);
        float x1, x2, y1, y2;
        x1 = Lerp(Grad(aaa, xf, yf     , zf), Grad(baa, xf - 1.0f, yf       , zf), u);
        x2 = Lerp(Grad(aba, xf, yf-1.0f, zf), Grad(bba, xf - 1.0f, yf - 1.0f, zf), u);
        y1 = Lerp(x1, x2, v);
        x1 = Lerp(Grad(aab, xf, yf     , zf-1.0f), Grad(bab, xf - 1.0f, yf       , zf - 1.0f), u);
        x2 = Lerp(Grad(abb, xf, yf-1.0f, zf-1.0f), Grad(bbb, xf - 1.0f, yf - 1.0f, zf - 1.0f), u);
        y2 = Lerp(x1, x2, v);
        return Lerp(y1,y2,w);
    }
private:
    static RTLIB_DEVICE auto Lerp(float x, float y, float t) -> float {
        //y * t+ (1.0f-t)*x
        //x + (y-x)*t
        return fmaf(y - x, t, x);
    }
    static RTLIB_DEVICE auto Grad(int hash, float x, float y, float z)-> float
    {
        switch (hash & 0xF)
        {
        case 0x0: return  x + y;
        case 0x1: return -x + y;
        case 0x2: return  x - y;
        case 0x3: return -x - y;
        case 0x4: return  x + z;
        case 0x5: return -x + z;
        case 0x6: return  x - z;
        case 0x7: return -x - z;
        case 0x8: return  y + z;
        case 0x9: return -y + z;
        case 0xA: return  y - z;
        case 0xB: return -y - z;
        case 0xC: return  y + x;
        case 0xD: return -y + z;
        case 0xE: return  y - x;
        case 0xF: return -y - z;
        default: return 0.0f; // never happens
        }
    }
    static RTLIB_DEVICE auto Fade(float t) -> float {
        return t * t * t * (t * (t * 6.0f - 15.0f) + 10.0f);
    }
};
namespace rtlib = RTLib::Backends::Cuda::Math;
extern "C" __global__ void randomKernel(unsigned int* seedBuffer, uchar4* outBuffer, int width, int height){
   int i = blockIdx.x * blockDim.x + threadIdx.x;
   int j = blockIdx.y * blockDim.y + threadIdx.y;
   if (i < width && j < height) {
       unsigned int seed = seedBuffer[j * width + i];
       auto rng  = rtlib::Xorshift32(seed);
       auto col0 = 1.0f*make_float3((Perline::Generate(i / 512.0f, j / 512.0f, 0)) + 1.0f) * 0.5f;
       auto col1 = 2.0f*make_float3((Perline::Generate(i / 256.0f, j / 256.0f, 0)) + 1.0f) * 0.5f;
       auto col2 = 4.0f*make_float3((Perline::Generate(i / 128.0f, j / 128.0f, 0)) + 1.0f) * 0.5f;
       auto col3 = 8.0f*make_float3((Perline::Generate(i /  64.0f, j /  64.0f, 0)) + 1.0f) * 0.5f;
       auto col  = (col0+col1 + col2 + col3) / 15.0f;
       outBuffer[j*width+i] = make_uchar4(col.x*255,col.y*255,col.z*255,255);
       seedBuffer[j*width+i] = rng.m_seed;
   }
}