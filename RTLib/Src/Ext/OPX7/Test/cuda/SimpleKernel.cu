#include "hip/hip_runtime.h"
#define __HIPCC__
#include "SimpleKernel.h"
struct HitRecordUserData
{
    float3 radiance;
    float3 throughPut;
    float3 bsdfVal;
    float  bsdfPdf;
    unsigned int depth;
};
extern "C" {
    __constant__ Params params;
}
namespace rtlib = RTLib::Ext::CUDA::Math;
extern "C" __global__ void       __raygen__rg() {
    const uint3 idx = optixGetLaunchIndex();
    const uint3 dim = optixGetLaunchDimensions();
    auto* rgData = reinterpret_cast<RayGenData*>(optixGetSbtDataPointer());
    const auto seed = params.seedBuffer[params.width * idx.y + idx.x];
    float3 result = params.accumBuffer[params.width * idx.y + idx.x]*static_cast<float>(params.samplesForAccum);
    BasicHitRecord<HitRecordUserData> hrec; 
    hrec.seed = seed;

    float3 color = make_float3(0.0f);
    rtlib::Xorshift32 xor32(hrec.seed);

    const auto gitter = rtlib::random_float2(xor32);
    const float2 d = make_float2(
        (2.0f * static_cast<float>(idx.x + gitter.x) / static_cast<float>(dim.x)) - 1.0,
        (2.0f * static_cast<float>(idx.y + gitter.y) / static_cast<float>(dim.y)) - 1.0);

    hrec.rayOrigin    = rgData->GetRayOrigin();
    hrec.rayDirection = rgData->GetRayDirection(d);
    hrec.rayDistance  = 0.0f;
    hrec.cosine       = 0.0f;
    hrec.seed         = xor32.m_seed;
    hrec.flags        = 0;

    hrec.userData.throughPut = make_float3(1.0f);
    hrec.userData.radiance   = make_float3(0.0f);
    hrec.userData.bsdfVal    = make_float3(0.0f);
    hrec.userData.bsdfPdf    = 0.0f;
    hrec.userData.depth      = 0;
    while (true) {
        TraceRadiance(params.gasHandle, hrec.rayOrigin, hrec.rayDirection, 0.01f, 1.0e20f, hrec);
        color += hrec.userData.radiance;
        ++hrec.userData.depth;

        if (isnan(hrec.rayDirection.x)|| isnan(hrec.rayDirection.y)|| isnan(hrec.rayDirection.z)) {
            printf("error\n");
            break;
        }

        if ((hrec.flags & HIT_RECORD_FLAG_FINISH) || (hrec.userData.depth > 10)) {
            break;
        }
    }
    
    result += color;
    result /= static_cast<float>(params.samplesForLaunch + params.samplesForAccum);

    // printf("%f, %lf\n", texCoord.x, texCoord.y);
    params.accumBuffer[params.width * idx.y + idx.x] = result;
    params.frameBuffer[params.width * idx.y + idx.x] = rtlib::rgba_to_srgb(make_uchar4(static_cast<unsigned char>(255.99 * result.x), static_cast<unsigned char>(255.99 * result.y), static_cast<unsigned char>(255.99 * result.z), 255));
    params.seedBuffer [params.width * idx.y + idx.x] = hrec.seed;
}
extern "C" __global__ void       __miss__radiance() {
    auto* hrec = BasicHitRecord<HitRecordUserData>::GetGlobalPointer();
    auto* msData = reinterpret_cast<MissData*>(optixGetSbtDataPointer());

    hrec->SetGlobalRayOrigin(optixGetWorldRayOrigin());
    hrec->SetGlobalRayDirAndTmax(make_float4(optixGetWorldRayDirection(), optixGetRayTmax()));

    hrec->cosine = 0.0f;
    hrec->flags |= HIT_RECORD_FLAG_FINISH;

    hrec->userData.radiance = hrec->userData.throughPut * make_float3(msData->bgColor.x, msData->bgColor.y, msData->bgColor.z);
    hrec->userData.bsdfVal  = make_float3(1.0f);
}
extern "C" __global__ void __closesthit__radiance() {
    auto* hrec   = BasicHitRecord<HitRecordUserData>::GetGlobalPointer();
    auto* hgData = reinterpret_cast<HitgroupData*>(optixGetSbtDataPointer());
    auto primitiveId = optixGetPrimitiveIndex();
    auto uv = optixGetTriangleBarycentrics();

    auto distance  = optixGetRayTmax();
    auto position  = optixGetWorldRayOrigin() + distance * optixGetWorldRayDirection();
    auto texCrd    = hgData->GetTexCrd(uv,primitiveId);
    auto normal    = hgData->GetNormal(uv,primitiveId);

    auto diffuse   = hgData->SampleDiffuse(texCrd);
    //auto specualr = hgData->SampleSpecular(texCrd);
    auto emission  = hgData->SampleEmission(texCrd);

    auto xor32 = rtlib::Xorshift32(hrec->seed);
    auto   onb  = rtlib::ONB(normal);

    auto direction = rtlib::normalize(onb.local(rtlib::random_cosine_direction(xor32)));
    auto  cosine    = rtlib::dot(direction, normal);
    auto bsdfVal   = diffuse * RTLIB_M_INV_PI;
    auto bsdfPdf   = cosine  * RTLIB_M_INV_PI;

    hrec->SetGlobalRayOrigin(position + 0.01f * normal);
    hrec->SetGlobalRayDirAndTmax(make_float4(direction, distance));

    hrec->normal = normal;
    hrec->seed   = xor32.m_seed;
    hrec->cosine = cosine;
    hrec->flags |= HIT_RECORD_FLAG_COUNT_EMITTED;

    if (emission.x * emission.y * emission.z > 0.0f) {
        hrec->flags |= HIT_RECORD_FLAG_FINISH;
    }

    hrec->userData.radiance    = hrec->userData.throughPut * emission;
    hrec->userData.throughPut *= diffuse;
}
extern "C" __global__ void       __miss__occluded() {
    optixSetPayload_0(false);
}
extern "C" __global__ void __closesthit__occluded() {
    optixSetPayload_0(true);
}
extern "C" __global__ void     __anyhit__ah() {
    auto* hgData = reinterpret_cast<HitgroupData*>(optixGetSbtDataPointer());
}
extern "C" __global__ void __exception__ep() {
    auto code = optixGetExceptionCode();
    if (code == OPTIX_EXCEPTION_CODE_TRAVERSAL_DEPTH_EXCEEDED)
    {
        printf("%d\n", optixGetTransformListSize());
    }
}