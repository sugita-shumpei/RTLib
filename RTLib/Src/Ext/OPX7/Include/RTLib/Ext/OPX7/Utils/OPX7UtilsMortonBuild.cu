#include <hip/hip_runtime.h>
extern "C" __global__ void mortonBuildKernel(float* weightBuilding, unsigned int level, unsigned int nodesPerElement, unsigned int numNodes) {
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < numNodes){
        float* weightBuildingNodeStart  = weightBuilding + idx * nodesPerElement * numNodes;
        for (int i = 0;i<level-1;++i){
            unsigned int srcOffset = (__powf(4.0f,level  -i)-1)/3;
            unsigned int dstOffset = (__powf(4.0f,level-1-i)-1)/3;
            for (unsigned int code=0;code<powf(4.0f,level-i);++code)
            {
                weightBuildingNodeStart[dstOffset+(code>>2)]+= weightBuildingNodeStart[srcOffset+code];
            }
        }
        
    }
}